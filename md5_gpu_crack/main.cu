#include "hip/hip_runtime.h"
#include <python2.7/Python.h>
#include "md5.cu"

#define LOOP 100

__device__ const int DEVICE_LOOP = LOOP;
__device__ int global_success = 0;
__device__ char *haha;

__device__ void copyWord(unsigned char* bruteWord, unsigned char* answer, int wordSize) {
	int i;
	for (i = 0; i < wordSize; i++) {
		answer[i] = bruteWord[i];
	}
	answer[wordSize] = '\0';
}

__device__ void fillZero(unsigned char* word, int wordSize) {
	int i;
	for (i = 0; i < wordSize; i++) {
		word[i] = 0;
	}
	word[wordSize] = '\0';
}

__device__ int strsize(char *string) {
	int i = 0;
	while (string[i] != '\0') {
		i++;
	}
	return i;
}

__device__ void makeBruteWord(unsigned char* indexWord, unsigned char* bruteWord, char* string, int digit) {
	int i;
	for (i = 0; i < digit; i++) {
		bruteWord[i] = string[indexWord[i]];
	}
}

__device__ void fowardWord(unsigned char* indexWord, int digit, char* string, int stringSize,
		unsigned long long int increment) {

	int i = digit - 1;
	while (increment > 0 && i >= 0) {
		unsigned long long int add = increment + indexWord[i];
//		printf("[%d] indexWord[i]:%d fowardWord increment:%-15llu  wordMax:%d  stringSize:%d  add:%-10llu  addstringSize:%u\n", i,  indexWord[i], increment, wordMax, stringSize, add, add % stringSize);
		indexWord[i] = add % stringSize;
		increment = add / stringSize;
		i -= 1;
	}
}

__global__ void crack(char* string, int *digit, uint *h1, uint *h2, uint *h3, uint *h4,
		unsigned long long int* increments, int *hasFound, unsigned char* answer, unsigned char* checkWord) {
	// Get Thread Index
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int offset = blockDim.x * gridDim.x;
	unsigned long long int increment;

	// Declare
	unsigned char* indexWord = new unsigned char[*digit + 1];
	unsigned char* bruteWord = new unsigned char[*digit + 1];
	uint v1, v2, v3, v4;
	int loop = 0;
	int deviceLoopMax = DEVICE_LOOP;

	// Initialization
	increment = increments[idx];
	increments[idx] += DEVICE_LOOP;

	fillZero(indexWord, *digit);
	fillZero(bruteWord, *digit);

	fowardWord(indexWord, *digit, string, strsize(string), increment + idx);
//	makeBruteWord(indexWord, bruteWord, string, *digit);
//	copyWord(bruteWord, checkWord, *digit);

	while (global_success == 0 && loop <= deviceLoopMax) {

		makeBruteWord(indexWord, bruteWord, string, *digit);
//		if(idx == 0){
//			printf("[%u]DEVICE bruteWord:%s increment:%llu offset:%u\n", idx, bruteWord, increment, offset);
//		}
//		if (idx == 0 && increment % 5000000 == 0 && loop == deviceLoopMax - 1) {
//			printf("[%u]DEVICE bruteWord:%s increment:%llu\n", idx, bruteWord, increment);
//		}

		if (idx == 0 && increment % 5000 == 0 && loop == deviceLoopMax - 1) {
			copyWord(bruteWord, checkWord, *digit);
		}

//		bruteWord[0] = 'a';
//		bruteWord[1] = 'b';
//		bruteWord[2] = '1';
//		bruteWord[3] = '4';
//		bruteWord[4] = '%';
//		bruteWord[5] = 'P';
//		bruteWord[6] = '\0';

		md5_vfy(bruteWord, *digit, &v1, &v2, &v3, &v4);
		if (*h1 == v1 && *h2 == v2 && *h3 == v3 && *h4 == v4 && global_success == 0) {
			*hasFound = 1;
			global_success = 1;
			copyWord(bruteWord, answer, *digit);

//			printf("DEVICE Found:%s bruteWord:%s\n", answer, bruteWord);
//			printf("DEVICE %p %p %p %p\n", h1, h2, h3, h4);
//			printf("DEVICE h%u %u %u %u\n", *h1, *h2, *h3, *h4);
//			printf("DEVICE v%u %u %u %u\n", v1, v2, v3, v4);
			break;
		}

		fowardWord(indexWord, *digit, string, strsize(string), offset);
		loop += 1;
	}

	// Finish
	increments[idx] += offset;

	// Destory
	free(indexWord);
	free(bruteWord);
}

unsigned char* anderson_main(char* hash, int digit, const char* string, int N_BLOCK = 256, int N_THREAD = 1024,
		int display = 0) {
	int N_TOTAL = N_BLOCK * N_THREAD;

	// Declare Variables
	const unsigned long long int HOST_LOOP = pow(strlen(string), digit);
	uint h1, h2, h3, h4;
	unsigned long long int increments[N_TOTAL];
	unsigned char check[digit + 1];
	unsigned char* answer = (unsigned char*) malloc(digit + 1);
	int hasFound = 0;
	int loop = 0;
	int i;

	// Declare CUDA Variables
	unsigned char *dev_answer;
	unsigned char *dev_check;
	char *dev_string;
	int *dev_digit;
	int *dev_hasFound;
	uint *dev_h1;
	uint *dev_h2;
	uint *dev_h3;
	uint *dev_h4;
	unsigned long long int *dev_increments;

	// Hash Initialization
	md5_to_ints((unsigned char*) hash, &h1, &h2, &h3, &h4);

	// Init Increments
	for (i = 0; i < N_TOTAL; i++) {
		increments[i] = 0;
	}

	for (i = 0; i < digit; i++) {
		check[i] = 0;
	}
	check[digit] = '\0';

	// CUDA Memory Allocation
	hipMalloc((void**) &dev_string, sizeof(char) * (strlen(string) + 1));
	hipMalloc((void**) &dev_digit, sizeof(int));
	hipMalloc((void**) &dev_h1, sizeof(uint));
	hipMalloc((void**) &dev_h2, sizeof(uint));
	hipMalloc((void**) &dev_h3, sizeof(uint));
	hipMalloc((void**) &dev_h4, sizeof(uint));
	hipMalloc((void**) &dev_increments, sizeof(unsigned long long int) * N_TOTAL);
	hipMalloc((void**) &dev_hasFound, sizeof(int));
	hipMalloc((void**) &dev_answer, sizeof(unsigned char) * digit + 1);
	hipMalloc((void**) &dev_check, sizeof(unsigned char) * digit + 1);

	// CUDA Memory Copy
	hipMemcpy(dev_string, string, sizeof(char) * (strlen(string) + 1), hipMemcpyHostToDevice);
	hipMemcpy(dev_digit, &digit, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_h1, &h1, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_h2, &h2, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_h3, &h3, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_h4, &h4, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_increments, &increments, sizeof(unsigned long long int) * N_TOTAL, hipMemcpyHostToDevice);
	hipMemcpy(dev_hasFound, &hasFound, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_answer, &answer, sizeof(unsigned char) * (digit + 1), hipMemcpyHostToDevice);
	hipMemcpy(dev_check, &check, sizeof(unsigned char) * (digit + 1), hipMemcpyHostToDevice);

	// Crack!
	while (loop < HOST_LOOP && hasFound == 0) {
		crack<<<N_BLOCK, N_THREAD>>>(dev_string, dev_digit, dev_h1, dev_h2, dev_h3, dev_h4, dev_increments,
				dev_hasFound, dev_answer, dev_check);

		hipMemcpy(&hasFound, dev_hasFound, sizeof(int), hipMemcpyDeviceToHost);
		if (hasFound == 1) {
			hipMemcpy(answer, dev_answer, sizeof(unsigned char) * (digit + 1), hipMemcpyDeviceToHost);
			if(display == 1){
				printf("ANSWER: %s\n", answer);
			}
			break;
		}

		if (display == 1 && loop % 100 == 0) {
			hipMemcpy(&check, dev_check, sizeof(unsigned char) * (digit + 1), hipMemcpyDeviceToHost);
			printf("Progress: %s\n", check);
		}

		loop += 1;
	}

	// Destroy..
	hipFree(dev_increments);
	hipFree(dev_string);
	hipFree(dev_digit);
	hipFree(dev_h1);
	hipFree(dev_h2);
	hipFree(dev_h3);
	hipFree(dev_h4);
	hipFree(dev_hasFound);
	hipFree(dev_answer);
	hipFree(dev_check);

	free(dev_string);
	free(dev_digit);
	free(dev_h1);
	free(dev_h2);
	free(dev_h3);
	free(dev_h4);
	free(dev_hasFound);
	free(dev_answer);
	free(dev_check);
	return answer;
}

int main(int argc, char **argv) {
	if (argc < 3) {
		printf(" hash digit [num of blocks] [num of threads] [possible string]\n");
		return 1;
	}

	char* hash = argv[1];
	int digit = atoi(argv[2]);
	const char *string =
			"abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!\"#$%&\'()*+,-./:;<=>?@[\\]^_`{|}~\0";
	int blocks = 512;
	int threads = 1024;
        printf("argc: %d %s \n", argc, argv[4]);
	if (argc >= 4) {
		blocks = atoi(argv[3]);
	}
	if (argc >= 5) {
		threads = atoi(argv[4]);
	}

	if (argc >= 6) {
		string = argv[5];
	}
	
	printf("hash:%s digit:%d string:%s blocks:%d threads:%d\n", hash, digit, string, blocks, threads);

	anderson_main(hash, digit, string, blocks, threads, 1);
	free(hash);
	return 1;
}

