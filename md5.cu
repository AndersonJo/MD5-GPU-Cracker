
#include <hip/hip_runtime.h>
//
// MD5 routines (straight from Wikipedia's MD5 pseudocode description)
//

//////////////////////////////////////////////////////////////////////////////
/////////////       Ron Rivest's MD5 C Implementation       //////////////////
//////////////////////////////////////////////////////////////////////////////

/*
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */

/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }

__device__ void md5_vfy(unsigned char* data, uint length, uint *a1, uint *b1,
		uint *c1, uint *d1) {

	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;

	uint a = 0;
	uint b = 0;
	uint c = 0;
	uint d = 0;

	uint vals[14] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	int i = 0;

	for (i = 0; i < length; i++) {
		vals[i / 4] |= data[i] << ((i % 4) * 8);
	}
	vals[i / 4] |= 0x80 << ((i % 4) * 8);

	uint bitlen = length * 8;

#define in0  (vals[0])//x
#define in1  (vals[1])//y
#define in2  (vals[2])//z
#define in3  (vals[3])
#define in4  (vals[4])
#define in5  (vals[5])
#define in6  (vals[6])
#define in7  (vals[7])
#define in8  (vals[8])
#define in9  (vals[9])
#define in10 (vals[10])
#define in11 (vals[11])
#define in12 (vals[12])
#define in13 (vals[13])
#define in14 (bitlen) //w = bit length
#define in15 (0)

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

	/* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
	FF(a, b, c, d, in0, S11, 3614090360); /* 1 */
	FF(d, a, b, c, in1, S12, 3905402710); /* 2 */
	FF(c, d, a, b, in2, S13, 606105819); /* 3 */
	FF(b, c, d, a, in3, S14, 3250441966); /* 4 */
	FF(a, b, c, d, in4, S11, 4118548399); /* 5 */
	FF(d, a, b, c, in5, S12, 1200080426); /* 6 */
	FF(c, d, a, b, in6, S13, 2821735955); /* 7 */
	FF(b, c, d, a, in7, S14, 4249261313); /* 8 */
	FF(a, b, c, d, in8, S11, 1770035416); /* 9 */
	FF(d, a, b, c, in9, S12, 2336552879); /* 10 */
	FF(c, d, a, b, in10, S13, 4294925233); /* 11 */
	FF(b, c, d, a, in11, S14, 2304563134); /* 12 */
	FF(a, b, c, d, in12, S11, 1804603682); /* 13 */
	FF(d, a, b, c, in13, S12, 4254626195); /* 14 */
	FF(c, d, a, b, in14, S13, 2792965006); /* 15 */
	FF(b, c, d, a, in15, S14, 1236535329); /* 16 */

	/* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
	GG(a, b, c, d, in1, S21, 4129170786); /* 17 */
	GG(d, a, b, c, in6, S22, 3225465664); /* 18 */
	GG(c, d, a, b, in11, S23, 643717713); /* 19 */
	GG(b, c, d, a, in0, S24, 3921069994); /* 20 */
	GG(a, b, c, d, in5, S21, 3593408605); /* 21 */
	GG(d, a, b, c, in10, S22, 38016083); /* 22 */
	GG(c, d, a, b, in15, S23, 3634488961); /* 23 */
	GG(b, c, d, a, in4, S24, 3889429448); /* 24 */
	GG(a, b, c, d, in9, S21, 568446438); /* 25 */
	GG(d, a, b, c, in14, S22, 3275163606); /* 26 */
	GG(c, d, a, b, in3, S23, 4107603335); /* 27 */
	GG(b, c, d, a, in8, S24, 1163531501); /* 28 */
	GG(a, b, c, d, in13, S21, 2850285829); /* 29 */
	GG(d, a, b, c, in2, S22, 4243563512); /* 30 */
	GG(c, d, a, b, in7, S23, 1735328473); /* 31 */
	GG(b, c, d, a, in12, S24, 2368359562); /* 32 */

	/* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
	HH(a, b, c, d, in5, S31, 4294588738); /* 33 */
	HH(d, a, b, c, in8, S32, 2272392833); /* 34 */
	HH(c, d, a, b, in11, S33, 1839030562); /* 35 */
	HH(b, c, d, a, in14, S34, 4259657740); /* 36 */
	HH(a, b, c, d, in1, S31, 2763975236); /* 37 */
	HH(d, a, b, c, in4, S32, 1272893353); /* 38 */
	HH(c, d, a, b, in7, S33, 4139469664); /* 39 */
	HH(b, c, d, a, in10, S34, 3200236656); /* 40 */
	HH(a, b, c, d, in13, S31, 681279174); /* 41 */
	HH(d, a, b, c, in0, S32, 3936430074); /* 42 */
	HH(c, d, a, b, in3, S33, 3572445317); /* 43 */
	HH(b, c, d, a, in6, S34, 76029189); /* 44 */
	HH(a, b, c, d, in9, S31, 3654602809); /* 45 */
	HH(d, a, b, c, in12, S32, 3873151461); /* 46 */
	HH(c, d, a, b, in15, S33, 530742520); /* 47 */
	HH(b, c, d, a, in2, S34, 3299628645); /* 48 */

	/* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
	II(a, b, c, d, in0, S41, 4096336452); /* 49 */
	II(d, a, b, c, in7, S42, 1126891415); /* 50 */
	II(c, d, a, b, in14, S43, 2878612391); /* 51 */
	II(b, c, d, a, in5, S44, 4237533241); /* 52 */
	II(a, b, c, d, in12, S41, 1700485571); /* 53 */
	II(d, a, b, c, in3, S42, 2399980690); /* 54 */
	II(c, d, a, b, in10, S43, 4293915773); /* 55 */
	II(b, c, d, a, in1, S44, 2240044497); /* 56 */
	II(a, b, c, d, in8, S41, 1873313359); /* 57 */
	II(d, a, b, c, in15, S42, 4264355552); /* 58 */
	II(c, d, a, b, in6, S43, 2734768916); /* 59 */
	II(b, c, d, a, in13, S44, 1309151649); /* 60 */
	II(a, b, c, d, in4, S41, 4149444226); /* 61 */
	II(d, a, b, c, in11, S42, 3174756917); /* 62 */
	II(c, d, a, b, in2, S43, 718787259); /* 63 */
	II(b, c, d, a, in9, S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

	*a1 = a;
	*b1 = b;
	*c1 = c;
	*d1 = d;
}

uint unhex(unsigned char x) {
	if (x <= 'F' && x >= 'A') {
		return (uint) (x - 'A' + 10);
	} else if (x <= 'f' && x >= 'a') {
		return (uint) (x - 'a' + 10);
	} else if (x <= '9' && x >= '0') {
		return (uint) (x - '0');
	}
	return 0;
}

void md5_to_ints(unsigned char* md5, uint *r0, uint *r1, uint *r2, uint *r3) {
	uint v0 = 0, v1 = 0, v2 = 0, v3 = 0;
	int i = 0;
	for (i = 0; i < 32; i += 2) {
		uint first = unhex(md5[i]);
		uint second = unhex(md5[i + 1]);
		uint both = first * 16 + second;
		both = both << 24;
		if (i < 8) {
			v0 = (v0 >> 8) | both;
		} else if (i < 16) {
			v1 = (v1 >> 8) | both;
		} else if (i < 24) {
			v2 = (v2 >> 8) | both;
		} else if (i < 32) {
			v3 = (v3 >> 8) | both;
		}
	}

	*r0 = v0;
	*r1 = v1;
	*r2 = v2;
	*r3 = v3;
}

