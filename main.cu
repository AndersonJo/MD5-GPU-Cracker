#include "hip/hip_runtime.h"
// Cracking MD5 with Nvidia GPU
// @Developer : Anderson Jo
// @email: a141890@gmail.com
// @website: http://andersonjo.github.io
// @copyrights: Use this library as you wish provided that
//              it is identified as "Made By Anderson Jo".
//              Do not remove developer name, email, website, and copyright.

#include <stdio.h>
#include <stdlib.h>
#include "md5.cu"
#define N (1024*33)

#define MAX_BRUTE_STRING_LENGTH 14
#define MD5_HASH_LENGTH 32

//Performance:
#define BLOCKS 65535
#define THREADS_PER_BLOCK 1024

__device__ void initWord(unsigned char* word, int wordMax, char* string, int stringSize, int tid) {
	int t = tid;
	int i;
	printf("initWord - wordMax:%d, \n", wordMax);
	for (i = 0; i < wordMax; i++) {
		word[i] = 0;
	}

	for (i = 0; i < wordMax; i++) {
		word[i]

	}

}

__global__ void searchHashWord(char* string, int* stringLength, int* wordMax, uint* h1, uint* h2, uint* h3, uint* h4) {
	uint v1, v2, v3, v4;

	printf("searchHashWord - stringLength:%d \n", stringLength);
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	unsigned char* word = new unsigned char[*wordMax];

	initWord(word, *wordMax, string, *stringLength, tid);

	if (tid >= (1024 - 1)) {
		printf("thread ID: %u\n", tid);
		printf("string %s %d\n", string);
	}

//	unsigned char* x = (unsigned char*) "ab14%P";
//	unsigned char* word[6];
//	md5_vfy(x, 6, &v1, &v2, &v3, &v4);

//	printf("%p %p %p %p\n", h1, h2, h3, h4);
//	printf("h%u %u %u %u\n", *h1, *h2, *h3, *h4);
//	printf("v%u %u %u %u\n", v1, v2, v3, v4);
}

int main(int argc, char **argv) {
	const char* string =
			"!\"#$%&\'()*+,-./0123456789:;<=>?@ABCDEFGHIJKLMNOPQRSTUVWXYZ[\\]^_`abcdefghijklmnopqrstuvwxyz{|}~";
	const int stringLength = strlen(string);
	const int wordMax = 30;

	unsigned char hash[MD5_HASH_LENGTH];
	memcpy(hash, "fe5f329d483283b7d03b03fc1e48e90c", MD5_HASH_LENGTH);

	// Get Unsigned Integers of Hash Key
	uint h1, h2, h3, h4;
	uint* dev_h1;
	uint* dev_h2;
	uint* dev_h3;
	uint* dev_h4;

	char* dev_string;
	int* dev_stringLength;
	int* dev_wordMax;

	md5_to_ints(hash, &h1, &h2, &h3, &h4);
	printf("%u %u %u %u\n", h1, h2, h3, h4);

	hipMalloc((void**) &dev_h1, sizeof(uint));
	hipMalloc((void**) &dev_h2, sizeof(uint));
	hipMalloc((void**) &dev_h3, sizeof(uint));
	hipMalloc((void**) &dev_h4, sizeof(uint));
	hipMalloc((void**) &dev_string, strlen(string));
	hipMalloc((void**) &dev_stringLength, sizeof(int));
	hipMalloc((void**) &dev_wordMax, sizeof(int));

	hipMemcpy(dev_h1, &h1, 1 * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_h2, &h2, 1 * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_h3, &h3, 1 * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_h4, &h4, 1 * sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(dev_string, string, strlen(string), hipMemcpyHostToDevice);
	hipMemcpy(dev_stringLength, &stringLength, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_wordMax, &wordMax, sizeof(int), hipMemcpyHostToDevice);

	printf("%p %p %p %p\n", dev_h1, dev_h2, dev_h3, dev_h4);
	searchHashWord<<<1, 1>>>(dev_string, dev_stringLength, dev_wordMax, dev_h1, dev_h2, dev_h3, dev_h4);

	hipFree(dev_h1);
	hipFree(dev_h2);
	hipFree(dev_h3);
	hipFree(dev_h4);
	hipFree(dev_string);
	hipFree(dev_stringLength);
	hipFree(dev_wordMax);
}

